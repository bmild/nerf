
#include <hip/hip_runtime.h>
#include "stdio.h"
__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}
int main()
{
    int a, b, c;
    int *dev_c;
    a = 3;
    b = 4;
    hipMalloc((void **)&dev_c, sizeof(int));
    add<<<1, 1>>>(a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d is %d\n", a, b, c);
    hipFree(dev_c);
    return 0;
}